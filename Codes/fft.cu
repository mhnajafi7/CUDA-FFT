#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!
//This code created by Mohammad H Najafi in June 2023

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// This kernel performs the bit-reversal permutation necessary for the radix-2 algorithm.
__global__ void radix2reorder(float* x_r_d,float* x_i_d,const unsigned int N, unsigned int M)
{
    int thrds = ( gridDim.x * gridDim.y * bz + gridDim.x * by + bx) * blockDim.x  + tx;
	unsigned int ic = thrds;
	unsigned int temp = 0;
	float R_temp[2];
	float I_temp[2];

	// Perform bit-reversal permutation
    for (int k = 0; k < 32; k++) {
        temp <<= 1;        
        temp |= (ic & 1); 
        ic >>= 1;                
    }
    ic = temp >> (32 - M);
	
	if(thrds < ic)
	{
		// Swap elements in the input signal
		I_temp[0] = x_i_d[ic];
		R_temp[0] = x_r_d[ic];
		I_temp[1] = x_i_d[thrds];
		R_temp[1] = x_r_d[thrds];
		x_i_d[thrds] = I_temp[0];
		x_r_d[thrds] = R_temp[0];
		x_i_d[ic] = I_temp[1];
		x_r_d[ic] = R_temp[1];
	}
}
// This kernel performs the butterfly operations for each stage of the radix-2 FFT.
__global__ void radix2(float* x_r_d, float* x_i_d ,const unsigned int N, unsigned int M) 
{
    int thrds = ( gridDim.x * gridDim.y * bz + gridDim.x * by + bx) * blockDim.x  + tx;
	float R_temp[2],I_temp[2];	
	unsigned int a = thrds + (thrds/M) * M;
	unsigned int b = thrds + (thrds/M) * M + M;

	I_temp[0] = x_i_d[a];
	R_temp[0] = x_r_d[a];
	I_temp[1] = x_i_d[b];
	R_temp[1] = x_r_d[b];	

	// Compute the angle for the butterfly operation
	float angle = - 2 * PI * ( (N/(M * 2)) * thrds  - (N/2) * (thrds/M) ) / N;
	float m = cos(angle);
	float n = sin(angle);
	
	// Perform the butterfly operation
	x_i_d[ a ] = I_temp[0] + R_temp[1] * n + I_temp[1] * m;
	x_r_d[ a ] = R_temp[0] + R_temp[1] * m - I_temp[1] * n;
	x_r_d[ b ] = R_temp[0] - R_temp[1] * m + I_temp[1] * n;
	x_i_d[ b ] = I_temp[0] - R_temp[1] * n - I_temp[1] * m;				



}

// This kernel performs the bit-reversal permutation necessary for the radix-4 algorithm.
__global__ void radix4reorder(float* x_r_d,float* x_i_d,const unsigned int N, unsigned int M)
{
    int thrds = ( gridDim.x * gridDim.y * bz + gridDim.x * by + bx) * blockDim.x  + tx;
	unsigned int ic = thrds;	
	unsigned int temp = 0;
	float R_temp[2];
	float I_temp[2];

	// Perform bit-reversal permutation
    for (int k = 0; k < 32; k += 2) {
        unsigned int bit1 = (ic >> k) & 0x01;
        unsigned int bit2 = (ic >> (k + 1)) & 0x01;
        temp |= (bit2 << (31 - k));
        temp |= (bit1 << (30 - k));
    }
    ic = temp >> (32 - M);
    		
	if(thrds < ic)
	{
		// Swap elements in the input signal
		I_temp[0] = x_i_d[ic];
		R_temp[0] = x_r_d[ic];
		I_temp[1] = x_i_d[thrds];
		R_temp[1] = x_r_d[thrds];
		x_i_d[thrds] = I_temp[0];
		x_r_d[thrds] = R_temp[0];
		x_i_d[ic] = I_temp[1];
		x_r_d[ic] = R_temp[1];
	}
}
// This kernel performs the butterfly operations for each stage of the radix-4 FFT.
__global__ void radix4(float* x_r_d, float* x_i_d, const unsigned int N, const unsigned int M) 
{
    int thrds = ( gridDim.x * gridDim.y * bz + gridDim.x * by + bx) * blockDim.x  + tx;
	float R_temp[4],I_temp[4];	
	float Re[4] , Im[4];
	unsigned int a[4];
	a[ 0 ] = (thrds/M) * 4*M + thrds%M + 0*M;
	a[ 1 ] = (thrds/M) * 4*M + thrds%M + 1*M;
	a[ 2 ] = (thrds/M) * 4*M + thrds%M + 2*M;
	a[ 3 ] = (thrds/M) * 4*M + thrds%M + 3*M;
		
	I_temp[0] = x_i_d[a[0]];
	R_temp[0] = x_r_d[a[0]];
	I_temp[1] = x_i_d[a[1]];
	R_temp[1] = x_r_d[a[1]];
	I_temp[2] = x_i_d[a[2]];
	R_temp[2] = x_r_d[a[2]];
	I_temp[3] = x_i_d[a[3]];	
	R_temp[3] = x_r_d[a[3]];

	// Compute the angle for the butterfly operations
	float angle  = - 2 * PI * ( thrds%M ) / ( 4*M );	
	float m[3] , n[3];
	m[0] = cos(angle);
	m[1] = cos(angle * 2);
	m[2] = cos(angle * 3);
	n[0] = sin(angle);
	n[1] = sin(angle * 2);
	n[2] = sin(angle * 3);

	Im[0] = I_temp[0];
	Re[0] = R_temp[0];
	Im[1] = R_temp[1] * n[0] + I_temp[1] * m[0];
	Re[1] = R_temp[1] * m[0] - I_temp[1] * n[0];
	Im[2] = R_temp[2] * n[1] + I_temp[2] * m[1];
	Re[2] = R_temp[2] * m[1] - I_temp[2] * n[1];
	Im[3] = R_temp[3] * n[2] + I_temp[3] * m[2];
	Re[3] = R_temp[3] * m[2] - I_temp[3] * n[2];
		
	// Perform the butterfly operations
	x_i_d[a[0]] = Im[0] + Im[1] + Im[2] + Im[3];
	x_r_d[a[0]] = Re[0] + Re[1] + Re[2] + Re[3];
	x_i_d[a[1]] = Im[0] - Re[1] - Im[2] + Re[3];
	x_r_d[a[1]] = Re[0] + Im[1] - Re[2] - Im[3];
	x_i_d[a[2]] = Im[0] - Im[1] + Im[2] - Im[3];
	x_r_d[a[2]] = Re[0] - Re[1] + Re[2] - Re[3];
	x_i_d[a[3]] = Im[0] + Re[1] - Im[2] - Re[3];
	x_r_d[a[3]] = Re[0] - Im[1] - Re[2] + Im[3];
	
}



//-----------------------------------------------------------------------------
// This is the main function that performs the FFT on the input signal using the specified radix.
void gpuKernel(float* x_r_d, float* x_i_d, /*float* X_r_d, float* X_i_d,*/ const unsigned int N, const unsigned int M)
{
	// Perform bit-reversal permutation for the radix-2 or radix-4 algorithm
	if((M % 2 == 1))
	{	
		// Perform the radix-2 FFT
		radix2reorder<<< dim3(N / (1024 * 512), 512, 1), 1024 >>>(x_r_d, x_i_d, N, M);	
		for(int k = 1; k < N; k *= 2)
			radix2 <<< dim3(N / (1024 * 512), 32, 32), 256 >>>(x_r_d, x_i_d, N, k);		
			
	}else{
		// Perform the radix-4 FFT	
		radix4reorder<<< dim3(N / (512 * 512), 32, 32), 256 >>>(x_r_d, x_i_d, N, M);
		
		for (int k = 1; k < N; k *= 4)
			radix4 <<< dim3(N / (1024 * 1024), 32, 32), 256 >>>(x_r_d, x_i_d, N, k);			
	}
}
